#include "hip/hip_runtime.h"
#include <algorithm>
#include <vector>

#include "caffe/layers/faces_proposal_layer.hpp"

namespace caffe {
/*
template <typename Dtype>
__global__ void FacesProposalDoForward(const int nthreads,
    const Dtype* const bias_data, const int num, const int channels,
    const int top_height, const int top_width, Dtype* const top_data) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    const int c = (index / top_height / top_width) % channels;
    top_data[index] += bias_data[c];
  }
}
*/


template <typename Dtype>
void FacesProposalLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
#if 1
	Forward_cpu(bottom, top);
#else
	int face_count = 2;
	top[0]->Reshape(face_count, 5, 1, 1); // faces:face_countx5
	Dtype* top0 = top[0]->mutable_cpu_data();
	float tmp[10] = {
			0, 10, 20, 30, 40,
			0, 50, 60, 70, 80
		};
	caffe_copy(10, (Dtype*)tmp, top0);


	Dtype* top1 = top[1]->mutable_cpu_data();
	float fc = face_count;
	caffe_copy(1, (Dtype*)&fc, top1);
    	/*FacesProposalDoForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        	count, bias_data, num, channels,
        	top_height, top_width, top_data);
	*/
#endif
}


INSTANTIATE_LAYER_GPU_FUNCS(FacesProposalLayer);



}
